#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <assert.h>
#include "hip/hip_runtime.h"
#include "cudaBHSpaceModel.h"
#include "build_config.h"


#define NULL_BODY (-1)
#define LOCK (-2)

//#define printf(...)

// nbodiesd -> number of leaf nodes
__constant__ int nbodiesd;
// dtimed -> update interval
__constant__ float dtimed, dthfd, epssqd, itolsqd;
// leaf node information, transfer from CPU to GPU
__constant__ volatile float *leaf_node_mass, *leaf_node_posx, *leaf_node_posy;
__constant__ volatile float *leaf_node_velx, *leaf_node_vely, *leaf_node_accx, *leaf_node_accy;
// internal node information, generated in GPU, should be hipMalloc-ed
__device__ volatile int internal_node_num;    // <= nbodiesd
__constant__ volatile int *internal_node_child;   // 4 x nbodiesd
__constant__ volatile float *internal_node_mass;    // nbodiesd
__constant__ volatile float *internal_node_posx;    // nbodiesd
__constant__ volatile float *internal_node_posy;    // nbodiesd
__device__ volatile float radiusd;
__constant__ volatile int *countd, *startd, *sortd;


__global__
void InitializationKernel() {
    // the below steps are originally done in another kernel
    radiusd = 700.f;
    int k = 0; // root node placed at first place
    for (int i = 0; i < 4; i++)
        internal_node_child[4 * k + i] = NULL_BODY;
    internal_node_mass[k] = -1.0f;
    internal_node_posx[k] = 500.f;
    internal_node_posy[k] = 300.f;
    startd[k] = 0;
    internal_node_num = 0;  // internel_node_num refers to the index of the last internal node
}


__global__
void TreeBuildingKernel() {
    // totally 16 registers used
    register float radius = radiusd;
    register float rootx = internal_node_posx[0];
    register float rooty = internal_node_posy[0];
    register int inc = blockDim.x * gridDim.x;
    register int i = threadIdx.x + blockIdx.x * blockDim.x;
    register float px = 0.f;
    register float py = 0.f;
    register float r = 0.f;
    register int n = 0;  // root
    register int j = 0;
    register int locked = 0;
    register int patch = 999999999;
    register int internal_node_idx;
    register int k = 0;
    register float x;
    register float y;


    while (i < nbodiesd) {
        px = leaf_node_posx[i];
        py = leaf_node_posy[i];
        n = 0;  // root
        r = radius;
        j = 0;
        if (rootx < px) j = 1;
        if (rooty < py) j += 2;
        int child = internal_node_child[n * 4 + j];
        while (child >= nbodiesd) {
            n = child - nbodiesd;
            r *= 0.5f;
            j = 0;
            if (internal_node_posx[n] < px) j = 1;
            if (internal_node_posy[n] < py) j += 2;
            child = internal_node_child[n * 4 + j];
        }

        if (child != LOCK) {
            locked = n * 4 + j;
            if (child == atomicCAS((int *)&internal_node_child[locked], child, LOCK)) {
                if (child == NULL_BODY) { // if it is a null pointer, just insert the new body
                    internal_node_child[locked] = i;
                } else {  // there already is a body in this position
                    patch = 999999999;
                    do {
                        internal_node_idx = atomicAdd((int *)&internal_node_num, 1) + 1;
                        patch = min(patch, internal_node_idx);

                        x = (j & 1) * r;
                        y = ((j >> 1) & 1) * r;
                        r *= 0.5f;
                        internal_node_mass[internal_node_idx] = -1.0f;
                        startd[internal_node_idx] = -1;
                        x = internal_node_posx[internal_node_idx] = internal_node_posx[n] - r + x;
                        y = internal_node_posy[internal_node_idx] = internal_node_posy[n] - r + y;
                        for (k = 0; k < 4; k++) {
                            internal_node_child[internal_node_idx * 4 + k] = NULL_BODY;
                        }
                        if (patch != internal_node_idx) {
                            internal_node_child[n * 4 + j] = internal_node_idx + nbodiesd;
                        }

                        j = 0;
                        if (x < leaf_node_posx[child]) j = 1;
                        if (y < leaf_node_posy[child]) j += 2;
                        internal_node_child[internal_node_idx * 4 + j] = child;
                        n = internal_node_idx;
                        j = 0;
                        if (x < px) j = 1;
                        if (y < py) j += 2;
                        child = internal_node_child[n * 4 + j];
                    } while (child >= 0);
                    internal_node_child[n * 4 + j] = i;
                    __threadfence();  // push out subtree
                    internal_node_child[locked] = patch + nbodiesd;
                }
                i += inc;  // move on to next body
            }
        }
        __syncthreads();  // throttle
    }
}


__global__
void SummarizationKernel() {
    // totally 12 registers used
    register int last_internal_node = internal_node_num;
    register int inc = blockDim.x * gridDim.x;
    __shared__ volatile int cache[256 * 4];
    register int j;
    register int cnt;
    register float cm = 0.0f;
    register float px = 0.0f;
    register float py = 0.0f;
    register float m;
    register int i;
    register int child;

    // bottom-up
    register int k = last_internal_node - (threadIdx.x + blockIdx.x * blockDim.x);
    register int missing = 0;

    while (k >= 0) {
        cm = 0.0f;
        px = 0.0f;
        py = 0.0f;
        cnt = 0;
        j = 0;

        if (missing == 0) {
            for (i = 0; i < 4; i++) {
                child = internal_node_child[k * 4 + i];
                // comment: if this child is not null pointer (may be internal or leaf)
                if (child >= 0) {
                    if (i != j) {
                        // comment:
                        // move children to front (needed later for speed)
                        internal_node_child[k * 4 + i] = -1;
                        internal_node_child[k * 4 + j] = child;
                    }
                    cache[missing * 256 + threadIdx.x] = child;
                    m = 0.f;
                    if (child >= nbodiesd) {
                        // it is a internal node
                        m = internal_node_mass[child - nbodiesd];
                    } else {
                        // it is a leaf
                        m = leaf_node_mass[child];
                    }
                    missing++;
                    if (m >= 0.0f) {
                        missing--;
                        cm += m;
                        if (child >= nbodiesd) {
                            cnt += (countd[child - nbodiesd]);
                            // assert(cnt >= 0);
                            // it is a internal node
                            px += internal_node_posx[child - nbodiesd] * m;
                            py += internal_node_posy[child - nbodiesd] * m;
                        } else {
                            // it is a leaf node
                            px += leaf_node_posx[child] * m;
                            py += leaf_node_posy[child] * m;
                        }
                    }
                    j++;
                }
            }
            cnt += j;
            // assert(cnt >= 0);
        }

        if (missing != 0) {
            do {
                // poll missing children
                child = cache[(missing - 1) * 256 + threadIdx.x];
                m = internal_node_mass[child - nbodiesd];
                if (m >= 0.f) {
                    missing--;
                    cm += m;
                    if (child >= nbodiesd) {
                        cnt += (countd[child - nbodiesd]);
                        assert(cnt >= 0);
                        // it is a internal node
                        px += internal_node_posx[child - nbodiesd] * m;
                        py += internal_node_posy[child - nbodiesd] * m;
                    } else {
                        // it is a leaf node
                        px += leaf_node_posx[child] * m;
                        py += leaf_node_posy[child] * m;
                    }
                }
            } while ((m >= 0.f) && (missing != 0));
        }

        if (missing == 0) {
            // all children are ready, so store computed information
            countd[k] = cnt;  // These counts make kernel 4 much faster
            m = 1.0f / cm;
            internal_node_posx[k] = px * m;
            internal_node_posy[k] = py * m;
            __threadfence();  // make sure data are visible before setting mass
            internal_node_mass[k] = cm;
            k -= inc;  // move on to next cell
        }
    }
}


__global__
void SortKernel() {
    // totally 5 registers used
    register int i, k, child, inc, start;

    inc = blockDim.x * gridDim.x;
    k = threadIdx.x + blockIdx.x * blockDim.x;

    while (k <= internal_node_num) {
        // it concurrently places the bodies into an array such that the bodies appear in the same order in the array as they would during an in-order traversal of the octree
        start = startd[k];
        if (start >= 0) {
            for (i = 0; i < 4; i++) {
                child = internal_node_child[k * 4 + i];
                if (child >= nbodiesd) {
                    // child is a cell
                    startd[child - nbodiesd] = start;
                    start += countd[child - nbodiesd];
                } else if (child >= 0) {
                    // child is a body
                    sortd[start] = child;
                    __threadfence();
                    start++;
                }
            }
            k += inc;
        }
        __syncthreads();  // throttle
    }
    __syncthreads();
}

// The most obvious problem with our recursive implementation is high execution divergence
__device__
float2 CalculateForceOnLeafNode(int leaf_node) {
    // notice: float2 overloads + (part of cuda runtime)
    // depth is for calculating region size (radiusd / (2^depth)); the depth of root is 0
    // target_node is the index in internel_node_child

    // initialize a stack and push_back root
    // there is no null pointer in the stack (check before push_back)
    register int stack[1024];   // store the index of internal nodes and leaf nodes
    register int stack_idx = -1;
    // push_back
    stack[++stack_idx] = 0;  // depth; for calculating s
    stack[++stack_idx] = nbodiesd;

    // while (1) {
    //  if stack is not empty
    //    fetch one element
    //    check if (s/d < SD_TRESHOLD):
    //      yes -> add to ax and ay
    //      no -> push its four children onto stack
    // }

    register int node_idx = 0;;
    register int depth = 0;
    register float s = 0.f;
    register float distance = 0.f;
    register float dx = 0.f, dy = 0.f;
    register float px = 0.f, py = 0.f;
    register float ax = 0.f, ay = 0.f;
    volatile float *x_array, *y_array, *mass_array;
    register bool isleaf;
    register int k;

    while (stack_idx >= 0) {
        // pop
        node_idx = stack[stack_idx--];
        depth = stack[stack_idx--];
        isleaf = false;

        if (node_idx >= 0 && node_idx < nbodiesd) {
            x_array = leaf_node_posx;
            y_array = leaf_node_posy;
            mass_array = leaf_node_mass;
            isleaf = true;
        } else {
            node_idx -= nbodiesd;
            x_array = internal_node_posx;
            y_array = internal_node_posy;
            mass_array = internal_node_mass;
        }

        s = radiusd / (1 << depth);
        px = leaf_node_posx[leaf_node];
        py = leaf_node_posy[leaf_node];
        // if the target_node is a leaf node, simply calculate the force and return
        dx = x_array[node_idx] - px;
        dy = y_array[node_idx] - py;

        distance = dx * dx + dy * dy + epssqd;
        distance = rsqrtf(distance);
        distance = mass_array[node_idx] * distance * distance;

        if (((s / distance) < SD_TRESHOLD) || isleaf) {
            //add to ax and ay
            ax += dx * distance;
            ay += dy * distance;
        } else {
            for (k = 0; k < 4; k++) {
                if (internal_node_child[4 * node_idx + k] != NULL_BODY) {
                    stack[++stack_idx] = depth + 1;   // push_back depth first
                    stack[++stack_idx] = internal_node_child[4 * node_idx + k];
                }
            }
        }
    }
    __syncthreads();
    return make_float2(ax, ay);
}

__global__
void ForceCalculationKernel() {
    // totally 15 registers used
    register int inc = blockDim.x * gridDim.x;
    register int i = threadIdx.x + blockIdx.x * blockDim.x;
    register int k;  // actual point
    register float2 acceleration;

    while (i < nbodiesd) {
        k = sortd[i];
        // k = i;
        acceleration = CalculateForceOnLeafNode(k);
        leaf_node_accx[k] = acceleration.x;
        leaf_node_accy[k] = acceleration.y;
        i += inc;
    }
}


__global__
void IntegrationKernel() {
    // totally 6 registers used
    register int i, inc;
    register float dvelx, dvely;
    register float velhx, velhy;

    // iterate over all bodies assigned to thread
    inc = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < nbodiesd; i += inc) {
        // integrate
        dvelx = leaf_node_accx[i] * dthfd;
        dvely = leaf_node_accy[i] * dthfd;

        velhx = leaf_node_velx[i] + dvelx;
        velhy = leaf_node_vely[i] + dvely;

        leaf_node_posx[i] += velhx * dtimed;
        leaf_node_posy[i] += velhy * dtimed;

        leaf_node_velx[i] = velhx + dvelx;
        leaf_node_vely[i] = velhy + dvely;
    }
}


cudaBHSpaceModel::cudaBHSpaceModel(RectangleD bounds, std::vector<Object> &objects, Screen *screen)
    : SpaceModel(bounds, objects, screen) {
    this->tree = new QuadTree(this->bounds);
    if (this->tree == NULL) {
        printf("unable to initialize QuadTree in SpaceModel\n");
        return;
    }
}

void checkLimit() {
    size_t limit = 0;
    //hipDeviceGetLimit(&limit, hipLimitStackSize);
    //printf("hipLimitStackSize: %u\n", (unsigned)limit);
    //hipDeviceGetLimit(&limit, hipLimitPrintfFifoSize);
    //printf("hipLimitPrintfFifoSize: %u\n", (unsigned)limit);
    //hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
    //printf("hipLimitMallocHeapSize: %u\n", (unsigned)limit);

    limit = 9999;

    hipDeviceSetLimit(hipLimitStackSize, limit);
    //hipDeviceSetLimit(hipLimitPrintfFifoSize, limit);
    //hipDeviceSetLimit(hipLimitMallocHeapSize, limit);

    limit = 0;

    hipDeviceGetLimit(&limit, hipLimitStackSize);
    printf("New hipLimitStackSize: %u\n", (unsigned)limit);
    //hipDeviceGetLimit(&limit, hipLimitPrintfFifoSize);
    //printf("New hipLimitPrintfFifoSize: %u\n", (unsigned)limit);
    //hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
    //printf("New hipLimitMallocHeapSize: %u\n", (unsigned)limit);
}


void
cudaBHSpaceModel::update(GS_FLOAT dt) {
    size_t i;
#ifdef CONST_TIME
    dt = CONST_TIME;
#endif
    int blocks;
    int nbodies;
    float *mass, *posx, *posy, *velx, *vely;
    float *massl, *posxl, *posyl, *velxl, *velyl;
    int *countl, *startl, *sortl;
    float *leaf_node_accxl, *leaf_node_accyl;
    int *internal_node_childl;
    float *internal_node_massl, *internal_node_posxl, *internal_node_posyl;
    float dtime, dthf, epssq, itolsq;

    blocks = 15; // number of multiprocessor, specific to K40m
    nbodies = this->objects.size();

    // checkLimit();

    // for segregate information of objects into different array
    // on the host
    mass = (float *)malloc(sizeof(float) * nbodies);
    posx = (float *)malloc(sizeof(float) * nbodies);
    posy = (float *)malloc(sizeof(float) * nbodies);
    velx = (float *)malloc(sizeof(float) * nbodies);
    vely = (float *)malloc(sizeof(float) * nbodies);
    for (i = 0; i < nbodies; i++) {
        posx[i] = this->objects[i].position.x;
        posy[i] = this->objects[i].position.y;
        velx[i] = this->objects[i].speed.x;
        vely[i] = this->objects[i].speed.y;
        mass[i] = this->objects[i].mass;
    }

    // on the device
    hipMalloc((void **)&massl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&posxl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&posyl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&velxl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&velyl, sizeof(float) * (nbodies + 1));

    // copy from host to device
    hipMemcpy(massl, mass, sizeof(float) * nbodies, hipMemcpyHostToDevice);
    hipMemcpy(posxl, posx, sizeof(float) * nbodies, hipMemcpyHostToDevice);
    hipMemcpy(posyl, posy, sizeof(float) * nbodies, hipMemcpyHostToDevice);
    hipMemcpy(velxl, velx, sizeof(float) * nbodies, hipMemcpyHostToDevice);
    hipMemcpy(velyl, vely, sizeof(float) * nbodies, hipMemcpyHostToDevice);

    // copy from __device__ to __constant__ for acceleration
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_mass), &massl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_posx), &posxl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_posy), &posyl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_velx), &velxl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_vely), &velyl, sizeof(void *));

    // for counting how many nodes an internal node have
    hipMalloc((void **)&countl, sizeof(int) * (nbodies + 1));
    hipMemset(countl, 0, sizeof(int) * (nbodies + 1));
    hipMemcpyToSymbol(HIP_SYMBOL(countd), &countl, sizeof(void *));

    hipMalloc((void **)&startl, sizeof(int) * (nbodies + 1));
    hipMemcpyToSymbol(HIP_SYMBOL(startd), &startl, sizeof(void *));

    hipMalloc((void **)&sortl, sizeof(int) * (nbodies + 1));
    hipMemcpyToSymbol(HIP_SYMBOL(sortd), &sortl, sizeof(void *));

    // allocate leaf node acceleration information
    hipMalloc((void **)&leaf_node_accxl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&leaf_node_accyl, sizeof(float) * (nbodies + 1));

    // copy to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_accx), &leaf_node_accxl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_accy), &leaf_node_accyl, sizeof(void *));

    // allocate space for internal nodes
    // on the device
    hipMalloc((void **)&internal_node_childl, sizeof(int) * (nbodies + 1) * 4);
    hipMalloc((void **)&internal_node_massl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&internal_node_posxl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&internal_node_posyl, sizeof(float) * (nbodies + 1));

    // copy to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(internal_node_child), &internal_node_childl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(internal_node_mass), &internal_node_massl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(internal_node_posx), &internal_node_posxl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(internal_node_posy), &internal_node_posyl, sizeof(void *));

    // calculation factors
    dtime = dt;
    dthf = dtime * 0.5f;
    epssq = 0.05 * 0.05;  // EPSILON; for soothing
    itolsq = 1.0f / (0.5 * 0.5);

    // copy the address of the array to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(nbodiesd), &nbodies, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dtimed), &dtime, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(dthfd), &dthf, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(epssqd), &epssq, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(itolsqd), &itolsq, sizeof(float));

    InitializationKernel <<< 1, 1>>>();
    TreeBuildingKernel <<< blocks *8, 256>>>();
    SummarizationKernel <<< blocks *8, 256>>>();
    SortKernel <<< blocks *8, 256>>>();
    // printf("before ForceCalculationKernel\n");
    ForceCalculationKernel <<< blocks *8, 256>>>();
    IntegrationKernel <<< blocks *FACTOR6, 256>>>();

    // we only need to copy these four back into host
    hipMemcpy(posx, posxl, sizeof(float) * nbodies, hipMemcpyDeviceToHost);
    hipMemcpy(posy, posyl, sizeof(float) * nbodies, hipMemcpyDeviceToHost);
    hipMemcpy(velx, velxl, sizeof(float) * nbodies, hipMemcpyDeviceToHost);
    hipMemcpy(vely, velyl, sizeof(float) * nbodies, hipMemcpyDeviceToHost);

    // update leaf_node_posx, leaf_node_posy, leaf_node_velx, leaf_node_vely to objects array
    for (i = 0; i < nbodies; i++) {
        this->objects[i].position.x = posx[i];
        this->objects[i].position.y = posy[i];
        this->objects[i].speed.x = velx[i];
        this->objects[i].speed.y = vely[i];
    }

    remove_objects_outside_bounds();

    free(mass);
    free(posx);
    free(posy);
    free(velx);
    free(vely);

    hipFree(massl);
    hipFree(posxl);
    hipFree(posyl);
    hipFree(velxl);
    hipFree(velyl);
    hipFree(startl);
    hipFree(countl);
    hipFree(sortl);
    hipFree(leaf_node_accxl);
    hipFree(leaf_node_accyl);
    hipFree(internal_node_childl);
    hipFree(internal_node_massl);
    hipFree(internal_node_posxl);
    hipFree(internal_node_posyl);
}

cudaBHSpaceModel::~cudaBHSpaceModel() {
    delete this->tree;
}

