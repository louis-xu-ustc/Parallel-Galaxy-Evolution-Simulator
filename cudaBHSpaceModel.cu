#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include "hip/hip_runtime.h"
#include "cutil_math.h"
#include "cudaBHSpaceModel.h"
#include "build_config.h"


#define NULL_BODY (-1)
#define LOCK (-2)

//#define printf(...) 

// thread count for a block
#define THREADS2 1024
#define THREADS3 1024
#define THREADS4 256
#define THREADS5 256
#define THREADS6 512

// block count = factor * #SMs
#define FACTOR2 1
#define FACTOR3 1  /* must all be resident at the same time */
#define FACTOR4 1  /* must all be resident at the same time */
#define FACTOR5 5
#define FACTOR6 3

// nbodiesd -> number of leaf nodes
__constant__ int nbodiesd;
// dtimed -> update interval
__constant__ float dtimed, dthfd, epssqd, itolsqd;
// leaf node information, transfer from CPU to GPU
__constant__ volatile float *leaf_node_mass, *leaf_node_posx, *leaf_node_posy, *leaf_node_velx, *leaf_node_vely, *leaf_node_accx, *leaf_node_accy;
// internal node information, generated in GPU, should be hipMalloc-ed
__device__ volatile int32_t internal_node_num;    // <= nbodiesd
__constant__ volatile int *internal_node_child;   // 4 x nbodiesd
__constant__ volatile float *internal_node_mass;    // nbodiesd
__constant__ volatile float *internal_node_posx;    // nbodiesd
__constant__ volatile float *internal_node_posy;    // nbodiesd
__device__ volatile float radiusd;


__global__
void InitializationKernel() {
    // the below steps are originally done in another kernel
    radiusd = 700.f;
    int k = 0; // root node placed at first place
    for (int i = 0; i < 4; i++)
      internal_node_child[4 * k + i] = NULL_BODY;
    internal_node_mass[k] = -1.0f;
    internal_node_posx[k] = 500.f;
    internal_node_posy[k] = 300.f;
    internal_node_num = 0;  // internel_node_num refers to the index of the last internal node
}


__global__
void TreeBuildingKernel()
{
  float radius = radiusd;
  float rootx = internal_node_posx[0];
  float rooty = internal_node_posy[0];
  int inc = blockDim.x * gridDim.x;
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < nbodiesd) {
    float px = leaf_node_posx[i];
    float py = leaf_node_posy[i];
    int n = 0;  // root
    float r = radius;
    int j = 0;
    if (rootx < px) j = 1;
    if (rooty < py) j += 2;
    int child = internal_node_child[n*4+j];
    while (child >= nbodiesd) {
      n = child - nbodiesd;
      r *= 0.5f;
      j = 0;
      if (internal_node_posx[n] < px) j = 1;
      if (internal_node_posy[n] < py) j += 2;
      child = internal_node_child[n*4+j];
    }

    if (child != LOCK) {
      int locked = n*4+j;
      if (child == atomicCAS((int *)&internal_node_child[locked], child, LOCK)) {
        if (child == NULL_BODY) { // if it is a null pointer, just insert the new body
          internal_node_child[locked] = i;
        } else {  // there already is a body in this position
          int patch = 999999999;
          do {
            int internal_node_idx = atomicAdd((int *)&internal_node_num, 1) + 1;
            patch = min(patch, internal_node_idx);

            float x = (j & 1) * r;
            float y = ((j >> 1) & 1) * r;
            r *= 0.5f;
            internal_node_mass[internal_node_idx] = -1.0f;
            x = internal_node_posx[internal_node_idx] = internal_node_posx[n] - r + x;
            y = internal_node_posy[internal_node_idx] = internal_node_posy[n] - r + y;
            for (int k = 0; k < 4; k++) {
              internal_node_child[internal_node_idx*4+k] = NULL_BODY;
            }
            if (patch != internal_node_idx) {
              internal_node_child[n*4+j] = internal_node_idx + nbodiesd;
            }

            j = 0;
            if (x < leaf_node_posx[child]) j = 1;
            if (y < leaf_node_posy[child]) j += 2;
            internal_node_child[internal_node_idx*4+j] = child;
            n = internal_node_idx;
            j = 0;
            if (x < px) j = 1;
            if (y < py) j += 2;
            child = internal_node_child[n*4+j];
          } while (child >= 0);
          internal_node_child[n*4+j] = i;
          __threadfence();  // push out subtree
          internal_node_child[locked] = patch + nbodiesd;
        }
        i += inc;  // move on to next body
      }
    }
    __syncthreads();  // throttle
  }
}


__global__
void SummarizationKernel()
{
  int last_internal_node = internal_node_num;
  int inc = blockDim.x * gridDim.x;

  int k = last_internal_node - (threadIdx.x + blockIdx.x * blockDim.x);
  int missing = 0;
  while (k >= 0) {
    printf("this is thread %d\n", k);
    int mask = 0;
    float cm = 0.0f;
    float px = 0.0f;
    float py = 0.0f;
    float m;

    if (missing == 0) {
      for (int i = 0; i < 4; i++) {
        int child = internal_node_child[k*4+i];
        // comment: if this child is not null pointer (may be internal or leaf)
        if (child >= 0) {
	  float m = 0.f;
          if (child >= nbodiesd) {
            // it is a internal node
            m = internal_node_mass[child - nbodiesd];
          } else {
            // it is a leaf
            m = leaf_node_mass[child];
          }
          missing++;
          if (m >= 0.0f) {
            missing--;
            cm += m;
            if (child >= nbodiesd) {
              // it is a internal node
              px += internal_node_posx[child - nbodiesd] * m;
              py += internal_node_posy[child - nbodiesd] * m;
            } else {
              // it is a leaf node
              px += leaf_node_posx[child] * m;
              py += leaf_node_posy[child] * m;
            }
          } else {
            mask |= (1 << i);
          }
        }
      }
    }

    if (missing != 0) {
        for (int i = 0; i < 4; i++) {
        // poll missing children
        if ((mask & (1 << i)) == 0) {
          // it was not missing
          continue;
        }

        int child = internal_node_child[k*4+i];
        if (child < 0) {
          // when this children is null pointer, skip it
          continue;
        } else {
          // if it is leaf node, ignore it because it was computed already
          if (child >= nbodiesd) {
            // solve the children one by one
            while (1) {
	      printf("I can't get out\n");
              // it is an internal node
              m = internal_node_mass[child - nbodiesd];
              if (m < 0.0f) {
                // if the internal node is not ready; keep retrying
                printf("waiting for it\n");
		continue;
              } else {
                // the internal is ready
                missing--;
                cm += m;
                px += internal_node_posx[child - nbodiesd] * m;
                py += internal_node_posy[child - nbodiesd] * m;
                break;
              }
            }
          }
        }
      }
    }

    printf("missing = %d\n", missing);

    if (missing == 0) {
      // all children are ready, so store computed information
      // countd[k] = cnt;
      float m = 1.0f / cm;
      internal_node_posx[k] = px * m;
      internal_node_posy[k] = py * m;
      __threadfence();  // make sure data are visible before setting mass
      internal_node_mass[k] = cm;
      k -= inc;  // move on to next cell
    } else {
      printf("it should be missing = 0, dude!\n");
    }
    __syncthreads();
  }
}


__device__
float2 CalculateForceOnLeafNode(int leaf_node, int depth, int target_node) {
  // notice: float2 overloads + (part of cuda runtime)
  // depth is for calculating region size (radiusd / (2^depth)); the depth of root is 0
  // target_node is the index in internel_node_child

  // when target_node < 0, this is a null children
  if (target_node < 0) {
    // it is null children
    return make_float2(0.f, 0.f);
  }

  float ax = 0.f, ay = 0.f;
  // if the target_node is a leaf node, simply calculate the force and return
  if (target_node >= 0 && target_node < nbodiesd) {
    float px = leaf_node_posx[leaf_node];
    float py = leaf_node_posy[leaf_node];

    float dx = leaf_node_posx[target_node] - px;
    float dy = leaf_node_posy[target_node] - py;

    float distance = dx*dx + dy*dy + epssqd;
    distance = rsqrtf(distance);
    distance = leaf_node_mass[target_node] * distance * distance;
    ax += dx * distance;
    ay += dy * distance;
    return make_float2(ax, ay);
  }

  // otherwise, calculate s/d, where s is the size of the region (of the target_node) 
  // and d is the actual distance
  target_node -= nbodiesd;   // conversion
  float s = radiusd / std::pow(2, depth);
  float distance = 0.f;
  float px = leaf_node_posx[leaf_node];
  float py = leaf_node_posy[leaf_node];
  float dx = internal_node_posx[target_node] - px;
  float dy = internal_node_posy[target_node] - py;
  distance = dx*dx + dy*dy + epssqd;  // d is the actual distance
  distance = rsqrtf(distance);
  distance = internal_node_mass[target_node] * distance * distance;

  // if s/d < ? (SD_TRESHOLD), see the internal node as an object, calculate the force and return
  if ((s/distance) < SD_TRESHOLD) {
    ax += dx * distance;
    ay += dy * distance;
    return make_float2(ax, ay);
  } else {
    // if s/d >= ?, do the above recursively on every child of the target_node
    return CalculateForceOnLeafNode(leaf_node, depth+1, internal_node_child[target_node*4]) + \
    CalculateForceOnLeafNode(leaf_node, depth+1, internal_node_child[target_node*4 + 1]) +    \
    CalculateForceOnLeafNode(leaf_node, depth+1, internal_node_child[target_node*4 + 2]) +    \
    CalculateForceOnLeafNode(leaf_node, depth+1, internal_node_child[target_node*4 + 3]);
  }

  printf("you shouldn't be here\n");
  return make_float2(0.f, 0.f);
}

__global__
void ForceCalculationKernel()
{
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < nbodiesd; i += blockDim.x * gridDim.x) {
    // (target_node = nbodiesd) for signifying root node
    float2 acceleration = CalculateForceOnLeafNode(i, 0, nbodiesd);
    float acccx = acceleration.x;
    float acccy = acceleration.y;
    leaf_node_accx[i] = acccx;
    leaf_node_accy[i] = acccy;
  }
}


__global__
void IntegrationKernel()
{
  register int i, inc;
  register float dvelx, dvely;
  register float velhx, velhy;

  // iterate over all bodies assigned to thread
  inc = blockDim.x * gridDim.x;
  for (i = threadIdx.x + blockIdx.x * blockDim.x; i < nbodiesd; i += inc) {
    // integrate
    dvelx = leaf_node_accx[i] * dthfd;
    dvely = leaf_node_accy[i] * dthfd;

    velhx = leaf_node_velx[i] + dvelx;
    velhy = leaf_node_vely[i] + dvely;

    leaf_node_posx[i] += velhx * dtimed;
    leaf_node_posy[i] += velhy * dtimed;

    leaf_node_velx[i] = velhx + dvelx;
    leaf_node_vely[i] = velhy + dvely;
  }
}


cudaBHSpaceModel::cudaBHSpaceModel(RectangleD bounds, std::vector<Object> &objects, Screen *screen)
    : SpaceModel(bounds, objects, screen) {
    this->tree = new QuadTree(this->bounds);
    if (this->tree == NULL) {
        printf("unable to initialize QuadTree in SpaceModel\n");
        return;
    }
}

void
cudaBHSpaceModel::update(GS_FLOAT dt) {
    size_t i;
#ifdef CONST_TIME
    dt = CONST_TIME;
#endif
    int blocks = 15; // number of multiprocessor, specific to K40m
    int nbodies = this->objects.size();

    // for segregate information of objects into different array
    // on the host
    float *mass = (float *)malloc(sizeof(float) * nbodies);
    float *posx = (float *)malloc(sizeof(float) * nbodies);
    float *posy = (float *)malloc(sizeof(float) * nbodies);
    float *velx = (float *)malloc(sizeof(float) * nbodies);
    float *vely = (float *)malloc(sizeof(float) * nbodies);
    for (i = 0; i < nbodies; i++) {
        posx[i] = this->objects[i].position.x;
        posy[i] = this->objects[i].position.y;
        velx[i] = this->objects[i].speed.x;
        vely[i] = this->objects[i].speed.y;
        mass[i] = this->objects[i].mass;
    }

    // on the device
    float *massl, *posxl, *posyl, *velxl, *velyl;
    hipMalloc((void **)&massl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&posxl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&posyl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&velxl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&velyl, sizeof(float) * (nbodies + 1));

    // copy from host to device
    hipMemcpy(massl, mass, sizeof(float) * nbodies, hipMemcpyHostToDevice);
    hipMemcpy(posxl, posx, sizeof(float) * nbodies, hipMemcpyHostToDevice);
    hipMemcpy(posyl, posy, sizeof(float) * nbodies, hipMemcpyHostToDevice);
    hipMemcpy(velxl, velx, sizeof(float) * nbodies, hipMemcpyHostToDevice);
    hipMemcpy(velyl, vely, sizeof(float) * nbodies, hipMemcpyHostToDevice);

    // copy from __device__ to __constant__ for acceleration
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_mass), &massl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_posx), &posxl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_posy), &posyl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_velx), &velxl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_vely), &velyl, sizeof(void *));

    // allocate leaf node acceleration information
    float *leaf_node_accxl, *leaf_node_accyl;
    hipMalloc((void **)&leaf_node_accxl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&leaf_node_accyl, sizeof(float) * (nbodies + 1));

    // copy to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_accx), &leaf_node_accxl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_accy), &leaf_node_accyl, sizeof(void *));

    printf("before allocate space for internal nodes\n");
    // allocate space for internal nodes
    // on the device
    int *internal_node_childl;
    float *internal_node_massl, *internal_node_posxl, *internal_node_posyl; 
    hipMalloc((void**)&internal_node_childl, sizeof(int) * (nbodies + 1) * 4);
    hipMalloc((void**)&internal_node_massl, sizeof(float) * (nbodies + 1));
    hipMalloc((void**)&internal_node_posxl, sizeof(float) * (nbodies + 1));
    hipMalloc((void**)&internal_node_posyl, sizeof(float) * (nbodies + 1));

    // copy to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(internal_node_child), &internal_node_childl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(internal_node_mass), &internal_node_massl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(internal_node_posx), &internal_node_posxl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(internal_node_posy), &internal_node_posyl, sizeof(void *));

    // calculation factors
    float dtime, dthf, epssq, itolsq;
    dtime = dt;
    dthf = dtime * 0.5f;
    epssq = 0.05 * 0.05;  // EPSILON; for soothing
    itolsq = 1.0f / (0.5 * 0.5);

    printf("before copy the address of the array to constant memory\n");
    // copy the address of the array to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(nbodiesd), &nbodies, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dtimed), &dtime, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(dthfd), &dthf, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(epssqd), &epssq, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(itolsqd), &itolsq, sizeof(float));

    printf("before InitializationKernel\n");
    InitializationKernel <<< 1, 1>>>();
    printf("before TreeBuildingKernel\n");
    TreeBuildingKernel <<< blocks * FACTOR3, THREADS3>>>();
    printf("before SummarizationKernel\n");
    SummarizationKernel <<< 1, THREADS4>>>();
    printf("before ForceCalculationKernel\n");
    ForceCalculationKernel <<< blocks * FACTOR5, THREADS5>>>();
    printf("before IntegrationKernel\n");
    IntegrationKernel <<< blocks * FACTOR6, THREADS6>>>();
    printf("after IntegrationKernel\n");

    // we only need to copy these four back into host
    hipMemcpy(posx, posxl, sizeof(float) * nbodies, hipMemcpyDeviceToHost);
    hipMemcpy(posy, posyl, sizeof(float) * nbodies, hipMemcpyDeviceToHost);
    hipMemcpy(velx, velxl, sizeof(float) * nbodies, hipMemcpyDeviceToHost);
    hipMemcpy(vely, velyl, sizeof(float) * nbodies, hipMemcpyDeviceToHost);

    // update leaf_node_posx, leaf_node_posy, leaf_node_velx, leaf_node_vely to objects array
    for (i = 0; i < nbodies; i++) {
    	float posxx = posx[i];
    	float posyy = posy[i];
    	float velxx = velx[i];
    	float velyy = vely[i];
        this->objects[i].position.x = posxx;
        this->objects[i].position.y = posyy;
        this->objects[i].speed.x = velxx;
        this->objects[i].speed.y = velyy;
    }

    remove_objects_outside_bounds();

    free(mass);
    free(posx);
    free(posy);
    free(velx);
    free(vely);

    hipFree(massl);
    hipFree(posxl);
    hipFree(posyl);
    hipFree(velxl);
    hipFree(velyl);
    hipFree(leaf_node_accxl);
    hipFree(leaf_node_accyl);
    hipFree(internal_node_childl);
    hipFree(internal_node_massl);
    hipFree(internal_node_posxl);
    hipFree(internal_node_posyl);
}

cudaBHSpaceModel::~cudaBHSpaceModel() {
    delete this->tree;
}

