#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <assert.h>
#include "hip/hip_runtime.h"
#include "cutil_math.h"
#include "cudaBHSpaceModel.h"
#include "build_config.h"


#define NULL_BODY (-1)
#define LOCK (-2)

//#define printf(...)

#define THREADS2 1024
#define THREADS3 1024
#define THREADS4 256
#define THREADS5 256
#define THREADS6 512

#define FACTOR2 1
#define FACTOR3 1
#define FACTOR4 1
#define FACTOR5 5
#define FACTOR6 3


// nbodiesd -> number of leaf nodes
__constant__ int nbodiesd;
// dtimed -> update interval
__constant__ float dtimed, dthfd, epssqd, itolsqd;
// leaf node information, transfer from CPU to GPU
__constant__ volatile float *leaf_node_mass, *leaf_node_posx, *leaf_node_posy;
__constant__ volatile float *leaf_node_velx, *leaf_node_vely, *leaf_node_accx, *leaf_node_accy;
// internal node information, generated in GPU, should be hipMalloc-ed
__device__ volatile int internal_node_num;    // <= nbodiesd
__constant__ volatile int *internal_node_child;   // 4 x nbodiesd
__constant__ volatile float *internal_node_mass;    // nbodiesd
__constant__ volatile float *internal_node_posx;    // nbodiesd
__constant__ volatile float *internal_node_posy;    // nbodiesd
__device__ volatile float radiusd;
__constant__ volatile int *countd, *startd, *sortd;


__global__
void InitializationKernel() {
    // the below steps are originally done in another kernel
    radiusd = 700.f;
    int k = 0; // root node placed at first place
    for (int i = 0; i < 4; i++)
        internal_node_child[4 * k + i] = NULL_BODY;
    internal_node_mass[k] = -1.0f;
    internal_node_posx[k] = 500.f;
    internal_node_posy[k] = 300.f;
    startd[k] = 0;
    internal_node_num = 0;  // internel_node_num refers to the index of the last internal node
}


__global__
void TreeBuildingKernel() {
    float radius = radiusd;
    float rootx = internal_node_posx[0];
    float rooty = internal_node_posy[0];
    int inc = blockDim.x * gridDim.x;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    while (i < nbodiesd) {
        float px = leaf_node_posx[i];
        float py = leaf_node_posy[i];
        int n = 0;  // root
        float r = radius;
        int j = 0;
        if (rootx < px) j = 1;
        if (rooty < py) j += 2;
        int child = internal_node_child[n * 4 + j];
        while (child >= nbodiesd) {
            n = child - nbodiesd;
            r *= 0.5f;
            j = 0;
            if (internal_node_posx[n] < px) j = 1;
            if (internal_node_posy[n] < py) j += 2;
            child = internal_node_child[n * 4 + j];
        }

        if (child != LOCK) {
            int locked = n * 4 + j;
            if (child == atomicCAS((int *)&internal_node_child[locked], child, LOCK)) {
                if (child == NULL_BODY) { // if it is a null pointer, just insert the new body
                    internal_node_child[locked] = i;
                } else {  // there already is a body in this position
                    int patch = 999999999;
                    do {
                        int internal_node_idx = atomicAdd((int *)&internal_node_num, 1) + 1;
                        patch = min(patch, internal_node_idx);

                        float x = (j & 1) * r;
                        float y = ((j >> 1) & 1) * r;
                        r *= 0.5f;
                        internal_node_mass[internal_node_idx] = -1.0f;
                        startd[internal_node_idx] = -1;
                        x = internal_node_posx[internal_node_idx] = internal_node_posx[n] - r + x;
                        y = internal_node_posy[internal_node_idx] = internal_node_posy[n] - r + y;
                        for (int k = 0; k < 4; k++) {
                            internal_node_child[internal_node_idx * 4 + k] = NULL_BODY;
                        }
                        if (patch != internal_node_idx) {
                            internal_node_child[n * 4 + j] = internal_node_idx + nbodiesd;
                        }

                        j = 0;
                        if (x < leaf_node_posx[child]) j = 1;
                        if (y < leaf_node_posy[child]) j += 2;
                        internal_node_child[internal_node_idx * 4 + j] = child;
                        n = internal_node_idx;
                        j = 0;
                        if (x < px) j = 1;
                        if (y < py) j += 2;
                        child = internal_node_child[n * 4 + j];
                    } while (child >= 0);
                    internal_node_child[n * 4 + j] = i;
                    __threadfence();  // push out subtree
                    internal_node_child[locked] = patch + nbodiesd;
                }
                i += inc;  // move on to next body
            }
        }
        __syncthreads();  // throttle
    }
}


__global__
void SummarizationKernel() {
    int last_internal_node = internal_node_num;
    int inc = blockDim.x * gridDim.x;
    __shared__ volatile int cache[THREADS3 * 4];
    int j;
    int cnt;

    // bottom-up
    int k = last_internal_node - (threadIdx.x + blockIdx.x * blockDim.x);
    int missing = 0;
    while (k >= 0) {
        float cm = 0.0f;
        float px = 0.0f;
        float py = 0.0f;
        float m;
        cnt = 0;
        j = 0;

        if (missing == 0) {
            for (int i = 0; i < 4; i++) {
                int child = internal_node_child[k * 4 + i];
                // comment: if this child is not null pointer (may be internal or leaf)
                if (child >= 0) {
                    if (i != j) {
                        // comment:
                        // move children to front (needed later for speed)
                        internal_node_child[k * 4 + i] = -1;
                        internal_node_child[k * 4 + j] = child;
                    }
                    cache[missing * THREADS3 + threadIdx.x] = child;
                    float m = 0.f;
                    if (child >= nbodiesd) {
                        // it is a internal node
                        m = internal_node_mass[child - nbodiesd];
                    } else {
                        // it is a leaf
                        m = leaf_node_mass[child];
                    }
                    missing++;
                    if (m >= 0.0f) {
                        missing--;
                        cm += m;
                        if (child >= nbodiesd) {
                            cnt += (countd[child - nbodiesd]);
                            // assert(cnt >= 0);
                            // it is a internal node
                            px += internal_node_posx[child - nbodiesd] * m;
                            py += internal_node_posy[child - nbodiesd] * m;
                        } else {
                            // it is a leaf node
                            px += leaf_node_posx[child] * m;
                            py += leaf_node_posy[child] * m;
                        }
                    }
                    j++;
                }
            }
            cnt += j;
            // assert(cnt >= 0);
        }

        if (missing != 0) {
            do {
                // poll missing children
                int child = cache[(missing - 1) * THREADS3 + threadIdx.x];
                m = internal_node_mass[child - nbodiesd];
                if (m >= 0.f) {
                    missing--;
                    cm += m;
                    if (child >= nbodiesd) {
                        cnt += (countd[child - nbodiesd]);
                        assert(cnt >= 0);
                        // it is a internal node
                        px += internal_node_posx[child - nbodiesd] * m;
                        py += internal_node_posy[child - nbodiesd] * m;
                    } else {
                        // it is a leaf node
                        px += leaf_node_posx[child] * m;
                        py += leaf_node_posy[child] * m;
                    }
                }
            } while ((m >= 0.f) && (missing != 0));
        }

        if (missing == 0) {
            // all children are ready, so store computed information
            countd[k] = cnt;  // These counts make kernel 4 much faster
            float m = 1.0f / cm;
            internal_node_posx[k] = px * m;
            internal_node_posy[k] = py * m;
            __threadfence();  // make sure data are visible before setting mass
            internal_node_mass[k] = cm;
            k -= inc;  // move on to next cell
        }
    }
}


__global__
void SortKernel() {
    int i, k, child, inc, start;

    inc = blockDim.x * gridDim.x;
    k = threadIdx.x + blockIdx.x * blockDim.x;

    while (k <= internal_node_num) {
        // it concurrently places the bodies into an array such that the bodies appear in the same order in the array as they would during an in-order traversal of the octree
        start = startd[k];
        if (start >= 0) {
            for (i = 0; i < 4; i++) {
                child = internal_node_child[k * 4 + i];
                if (child >= nbodiesd) {
                    // child is a cell
                    startd[child - nbodiesd] = start;
                    start += countd[child - nbodiesd];
                } else if (child >= 0) {
                    // child is a body
                    sortd[start] = child;
                    __threadfence();
                    start++;
                }
            }
            k += inc;
        }
        __syncthreads();  // throttle
    }
    __syncthreads();
}

__device__ GS_INLINE float2 float2_add(float2 a, float2 b) {
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__ GS_INLINE float2 float2_sub(float2 a, float2 b) {
    return make_float2(a.x - b.x, a.y - b.y);
}

__device__ GS_INLINE float2 float2_mul(float2 a, float x) {
    return make_float2(a.x * x, a.y * x);
}

__device__ GS_INLINE float2 float2_zero() {
    return make_float2(0.f, 0.f);
}

__device__ GS_INLINE GS_FLOAT float2_length(float2 dr) {
    return sqrt((dr.x * dr.x) + (dr.y * dr.y));
}

__device__ GS_INLINE float2 calculate_force(float2 aPos, float2 bPos, float bMass) {
    register float2 dr = float2_sub(bPos, aPos);
    register GS_FLOAT r = float2_length(dr) + SOFT_CONST;
    register GS_FLOAT f = G_CONST * bMass / SQUARE(r);
    return float2_mul(dr, f/r);
}


// The most obvious problem with our recursive implementation is high execution divergence
__device__
float2 CalculateForceOnLeafNode(int leaf_node) {
    // notice: float2 overloads + (part of cuda runtime)
    // depth is for calculating region size (radiusd / (2^depth)); the depth of root is 0
    // target_node is the index in internel_node_child

    // initialize a stack and push_back root
    // there is no null pointer in the stack (check before push_back)
    int stack[1024];   // store the index of internal nodes and leaf nodes
    int stack_idx = -1;
    // push_back
    stack[++stack_idx] = 0;  // depth; for calculating s
    stack[++stack_idx] = nbodiesd;

    // while (1) {
    //  if stack is not empty
    //    fetch one element
    //    check if (s/d < SD_TRESHOLD):
    //      yes -> add to ax and ay
    //      no -> push its four children onto stack
    // }

    int node_idx = 0;
    int depth = 0;
    float s = 0.f;
    // float distance = 0.f;
    // float dx = 0.f, dy = 0.f;
    float px = 0.f, py = 0.f;
    // float ax = 0.f, ay = 0.f;

    // register float2 tarObjPos = make_float2(leaf_node_posx[leaf_node], leaf_node_posy[leaf_node]);
    // register float2 result = float2_zero();
    register float2 acc;
    // register GS_FLOAT s, d;

    volatile float *x_array, *y_array, *mass_array;
    while (stack_idx >= 0) {
        // pop
        node_idx = stack[stack_idx--];
        depth = stack[stack_idx--];
        bool isleaf = false;

        if (node_idx >= 0 && node_idx < nbodiesd) {
            x_array = leaf_node_posx;
            y_array = leaf_node_posy;
            mass_array = leaf_node_mass;
            isleaf = true;
        } else {
            node_idx -= nbodiesd;
            x_array = internal_node_posx;
            y_array = internal_node_posy;
            mass_array = internal_node_mass;
        }

        s = radiusd / (1 << depth);
        px = leaf_node_posx[leaf_node];
        py = leaf_node_posy[leaf_node];
        // if the target_node is a leaf node, simply calculate the force and return
        // dx = x_array[node_idx] - px;
        // dy = y_array[node_idx] - py;

        float2 leafPos = make_float2(px, py);
        float2 targetPos = make_float2(x_array[node_idx], y_array[node_idx]);
        float2 dr = float2_sub(leafPos, targetPos);
        GS_FLOAT distance = float2_length(dr);

        // distance = dx * dx + dy * dy + epssqd;
        // distance = rsqrtf(distance);
        // distance = mass_array[node_idx] * distance * distance;

        if (((s / distance) < SD_TRESHOLD) || isleaf) {
            //add to ax and ay
            acc = float2_add(acc, calculate_force(leafPos, targetPos, mass_array[node_idx]));
        } else {
            for (int k = 0; k < 4; k++) {
                if (internal_node_child[4 * node_idx + k] != NULL_BODY) {
                    stack[++stack_idx] = depth + 1;   // push_back depth first
                    stack[++stack_idx] = internal_node_child[4 * node_idx + k];
                }
            }
        }
    }
    __syncthreads();
    return acc;
}

__global__
void ForceCalculationKernel() {
    int inc = blockDim.x * gridDim.x;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    volatile int k;  // actual point
    float2 acceleration;
    while (i < nbodiesd) {
        k = sortd[i];
        // k = i;
        acceleration = CalculateForceOnLeafNode(k);
        leaf_node_accx[k] = acceleration.x;
        leaf_node_accy[k] = acceleration.y;
        i += inc;
    }
}


__global__
void IntegrationKernel() {
    register int i, inc;
    register float dvelx, dvely;
    register float velhx, velhy;

    // iterate over all bodies assigned to thread
    inc = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < nbodiesd; i += inc) {
        // integrate
        dvelx = leaf_node_accx[i] * dthfd;
        dvely = leaf_node_accy[i] * dthfd;

        velhx = leaf_node_velx[i] + dvelx;
        velhy = leaf_node_vely[i] + dvely;

        leaf_node_posx[i] += velhx * dtimed;
        leaf_node_posy[i] += velhy * dtimed;

        leaf_node_velx[i] = velhx + dvelx;
        leaf_node_vely[i] = velhy + dvely;
    }
}


cudaBHSpaceModel::cudaBHSpaceModel(RectangleD bounds, std::vector<Object> &objects, Screen *screen)
    : SpaceModel(bounds, objects, screen) {
    this->tree = new QuadTree(this->bounds);
    if (this->tree == NULL) {
        printf("unable to initialize QuadTree in SpaceModel\n");
        return;
    }
}

void checkLimit() {
    size_t limit = 0;
    //hipDeviceGetLimit(&limit, hipLimitStackSize);
    //printf("hipLimitStackSize: %u\n", (unsigned)limit);
    //hipDeviceGetLimit(&limit, hipLimitPrintfFifoSize);
    //printf("hipLimitPrintfFifoSize: %u\n", (unsigned)limit);
    //hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
    //printf("hipLimitMallocHeapSize: %u\n", (unsigned)limit);

    limit = 9999;

    hipDeviceSetLimit(hipLimitStackSize, limit);
    //hipDeviceSetLimit(hipLimitPrintfFifoSize, limit);
    //hipDeviceSetLimit(hipLimitMallocHeapSize, limit);

    limit = 0;

    hipDeviceGetLimit(&limit, hipLimitStackSize);
    printf("New hipLimitStackSize: %u\n", (unsigned)limit);
    //hipDeviceGetLimit(&limit, hipLimitPrintfFifoSize);
    //printf("New hipLimitPrintfFifoSize: %u\n", (unsigned)limit);
    //hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
    //printf("New hipLimitMallocHeapSize: %u\n", (unsigned)limit);
}


void
cudaBHSpaceModel::update(GS_FLOAT dt) {
    size_t i;
#ifdef CONST_TIME
    dt = CONST_TIME;
#endif
    int blocks;
    int nbodies;
    float *mass, *posx, *posy, *velx, *vely;
    float *massl, *posxl, *posyl, *velxl, *velyl;
    int *countl, *startl, *sortl;
    float *leaf_node_accxl, *leaf_node_accyl;
    int *internal_node_childl;
    float *internal_node_massl, *internal_node_posxl, *internal_node_posyl;
    float dtime, dthf, epssq, itolsq;

    blocks = 15; // number of multiprocessor, specific to K40m
    nbodies = this->objects.size();

    // checkLimit();

    // for segregate information of objects into different array
    // on the host
    mass = (float *)malloc(sizeof(float) * nbodies);
    posx = (float *)malloc(sizeof(float) * nbodies);
    posy = (float *)malloc(sizeof(float) * nbodies);
    velx = (float *)malloc(sizeof(float) * nbodies);
    vely = (float *)malloc(sizeof(float) * nbodies);
    for (i = 0; i < nbodies; i++) {
        posx[i] = this->objects[i].position.x;
        posy[i] = this->objects[i].position.y;
        velx[i] = this->objects[i].speed.x;
        vely[i] = this->objects[i].speed.y;
        mass[i] = this->objects[i].mass;
    }

    // on the device
    hipMalloc((void **)&massl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&posxl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&posyl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&velxl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&velyl, sizeof(float) * (nbodies + 1));

    // copy from host to device
    hipMemcpy(massl, mass, sizeof(float) * nbodies, hipMemcpyHostToDevice);
    hipMemcpy(posxl, posx, sizeof(float) * nbodies, hipMemcpyHostToDevice);
    hipMemcpy(posyl, posy, sizeof(float) * nbodies, hipMemcpyHostToDevice);
    hipMemcpy(velxl, velx, sizeof(float) * nbodies, hipMemcpyHostToDevice);
    hipMemcpy(velyl, vely, sizeof(float) * nbodies, hipMemcpyHostToDevice);

    // copy from __device__ to __constant__ for acceleration
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_mass), &massl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_posx), &posxl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_posy), &posyl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_velx), &velxl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_vely), &velyl, sizeof(void *));

    // for counting how many nodes an internal node have
    hipMalloc((void **)&countl, sizeof(int) * (nbodies + 1));
    hipMemset(countl, 0, sizeof(int) * (nbodies + 1));
    hipMemcpyToSymbol(HIP_SYMBOL(countd), &countl, sizeof(void *));

    hipMalloc((void **)&startl, sizeof(int) * (nbodies + 1));
    hipMemcpyToSymbol(HIP_SYMBOL(startd), &startl, sizeof(void *));

    hipMalloc((void **)&sortl, sizeof(int) * (nbodies + 1));
    hipMemcpyToSymbol(HIP_SYMBOL(sortd), &sortl, sizeof(void *));

    // allocate leaf node acceleration information
    hipMalloc((void **)&leaf_node_accxl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&leaf_node_accyl, sizeof(float) * (nbodies + 1));

    // copy to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_accx), &leaf_node_accxl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(leaf_node_accy), &leaf_node_accyl, sizeof(void *));

    // allocate space for internal nodes
    // on the device
    hipMalloc((void **)&internal_node_childl, sizeof(int) * (nbodies + 1) * 4);
    hipMalloc((void **)&internal_node_massl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&internal_node_posxl, sizeof(float) * (nbodies + 1));
    hipMalloc((void **)&internal_node_posyl, sizeof(float) * (nbodies + 1));

    // copy to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(internal_node_child), &internal_node_childl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(internal_node_mass), &internal_node_massl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(internal_node_posx), &internal_node_posxl, sizeof(void *));
    hipMemcpyToSymbol(HIP_SYMBOL(internal_node_posy), &internal_node_posyl, sizeof(void *));

    // calculation factors
    dtime = dt;
    dthf = dtime * 0.5f;
    epssq = 0.05 * 0.05;  // EPSILON; for soothing
    itolsq = 1.0f / (0.5 * 0.5);

    // copy the address of the array to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(nbodiesd), &nbodies, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dtimed), &dtime, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(dthfd), &dthf, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(epssqd), &epssq, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(itolsqd), &itolsq, sizeof(float));

    InitializationKernel <<< 1, 1>>>();
    TreeBuildingKernel <<< blocks *FACTOR2, THREADS2>>>();
    SummarizationKernel <<< blocks *FACTOR3, THREADS3>>>();
    SortKernel <<< blocks *FACTOR4, THREADS4>>>();
    // printf("before ForceCalculationKernel\n");
    ForceCalculationKernel <<< blocks *FACTOR5, THREADS5>>>();
    IntegrationKernel <<< blocks *FACTOR6, THREADS6>>>();

    // we only need to copy these four back into host
    hipMemcpy(posx, posxl, sizeof(float) * nbodies, hipMemcpyDeviceToHost);
    hipMemcpy(posy, posyl, sizeof(float) * nbodies, hipMemcpyDeviceToHost);
    hipMemcpy(velx, velxl, sizeof(float) * nbodies, hipMemcpyDeviceToHost);
    hipMemcpy(vely, velyl, sizeof(float) * nbodies, hipMemcpyDeviceToHost);

    // update leaf_node_posx, leaf_node_posy, leaf_node_velx, leaf_node_vely to objects array
    for (i = 0; i < nbodies; i++) {
        this->objects[i].position.x = posx[i];
        this->objects[i].position.y = posy[i];
        this->objects[i].speed.x = velx[i];
        this->objects[i].speed.y = vely[i];
    }

    remove_objects_outside_bounds();

    free(mass);
    free(posx);
    free(posy);
    free(velx);
    free(vely);

    hipFree(massl);
    hipFree(posxl);
    hipFree(posyl);
    hipFree(velxl);
    hipFree(velyl);
    hipFree(startl);
    hipFree(countl);
    hipFree(sortl);
    hipFree(leaf_node_accxl);
    hipFree(leaf_node_accyl);
    hipFree(internal_node_childl);
    hipFree(internal_node_massl);
    hipFree(internal_node_posxl);
    hipFree(internal_node_posyl);
}

cudaBHSpaceModel::~cudaBHSpaceModel() {
    delete this->tree;
}

